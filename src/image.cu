#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <cassert>
#include <utility>
#include <vector>


#include "image.hpp"


__global__ void gaussian_blur_horizontal(float* input, float* output, float* kernel,
                                       int width, int height, int kernel_size, int center) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        float sum = 0.0f;
        for (int k = 0; k < kernel_size; k++) {
            int dx = -center + k;
            int src_x = min(max(x + dx, 0), width - 1);
            sum += input[y * width + src_x] * kernel[k];
        }
        output[y * width + x] = sum;
    }
}

__global__ void gaussian_blur_vertical(float* input, float* output, float* kernel,
                                     int width, int height, int kernel_size, int center) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        float sum = 0.0f;
        for (int k = 0; k < kernel_size; k++) {
            int dy = -center + k;
            int src_y = min(max(y + dy, 0), height - 1);
            sum += input[src_y * width + x] * kernel[k];
        }
        output[y * width + x] = sum;
    }
}

Image gaussian_blur_cuda(const Image& img, float sigma) 
{
    assert(img.channels == 1);
    
    // 计算高斯核
    int size = std::ceil(6 * sigma);
    if (size % 2 == 0) size++;
    int center = size / 2;
    
    std::vector<float> h_kernel(size);
    float sum = 0.0f;
    for (int k = -size/2; k <= size/2; k++) {
        float val = std::exp(-(k*k) / (2*sigma*sigma));
        h_kernel[center + k] = val;
        sum += val;
    }
    for (int k = 0; k < size; k++) {
        h_kernel[k] /= sum;
    }
    
    // 分配设备内存
    float *d_input, *d_temp, *d_output, *d_kernel;
    hipMalloc(&d_input, img.width * img.height * sizeof(float));
    hipMalloc(&d_temp, img.width * img.height * sizeof(float));
    hipMalloc(&d_output, img.width * img.height * sizeof(float));
    hipMalloc(&d_kernel, size * sizeof(float));
    
    // 复制数据到设备
    hipMemcpy(d_input, img.data, img.width * img.height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel.data(), size * sizeof(float), hipMemcpyHostToDevice);
    
    // 设置grid和block大小
    dim3 block(16, 16);
    dim3 grid((img.width + block.x - 1) / block.x, 
              (img.height + block.y - 1) / block.y);
    
    // 执行水平和垂直方向的卷积
    gaussian_blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel,
                                            img.width, img.height, size, center);
    gaussian_blur_vertical<<<grid, block>>>(d_temp, d_output, d_kernel,
                                          img.width, img.height, size, center);
    
    // 创建输出图像
    Image result(img.width, img.height, 1);
    
    // 复制结果回主机
    hipMemcpy(result.data, d_output, img.width * img.height * sizeof(float), hipMemcpyDeviceToHost);
    
    // 清理设备内存
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_output);
    hipFree(d_kernel);
    
    return result;
} 