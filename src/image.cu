#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <cassert>
#include <utility>
#include <vector>

#include "sift.hpp"
#include "image.hpp"


__global__ void rgb_to_grayscale_kernel(const float* rgb, float* gray, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = y * width + x;
        float red = rgb[3 * idx];
        float green = rgb[3 * idx + 1];
        float blue = rgb[3 * idx + 2];
        gray[idx] = 0.299f * red + 0.587f * green + 0.114f * blue;
    }
}

Image rgb_to_grayscale_cuda(const Image& img) {
    assert(img.channels == 3);
    Image gray(img.width, img.height, 1);

    // Allocate device memory
    float *d_rgb, *d_gray;
    size_t rgb_size = img.width * img.height * 3 * sizeof(float);
    size_t gray_size = img.width * img.height * sizeof(float);
    
    hipMalloc(&d_rgb, rgb_size);
    hipMalloc(&d_gray, gray_size);

    // Copy RGB data to device
    hipMemcpy(d_rgb, img.data, rgb_size, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 block(16, 16);
    dim3 grid((img.width + block.x - 1) / block.x,
              (img.height + block.y - 1) / block.y);

    // Launch kernel
    rgb_to_grayscale_kernel<<<grid, block>>>(d_rgb, d_gray, img.width, img.height);

    // Copy result back to host
    hipMemcpy(gray.data, d_gray, gray_size, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_rgb);
    hipFree(d_gray);

    return gray;
}


__global__ void gaussian_blur_horizontal(float* input, float* output, float* kernel,
                                       int width, int height, int kernel_size, int center) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        float sum = 0.0f;
        for (int k = 0; k < kernel_size; k++) {
            int dx = -center + k;
            int src_x = min(max(x + dx, 0), width - 1);
            sum += input[y * width + src_x] * kernel[k];
        }
        output[y * width + x] = sum;
    }
}

__global__ void gaussian_blur_vertical(float* input, float* output, float* kernel,
                                     int width, int height, int kernel_size, int center) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        float sum = 0.0f;
        for (int k = 0; k < kernel_size; k++) {
            int dy = -center + k;
            int src_y = min(max(y + dy, 0), height - 1);
            sum += input[src_y * width + x] * kernel[k];
        }
        output[y * width + x] = sum;
    }
}

Image gaussian_blur_cuda(const Image& img, float sigma) 
{
    assert(img.channels == 1);
    
    // 计算高斯核
    int size = std::ceil(6 * sigma);
    if (size % 2 == 0) size++;
    int center = size / 2;
    
    std::vector<float> h_kernel(size);
    float sum = 0.0f;
    for (int k = -size/2; k <= size/2; k++) {
        float val = std::exp(-(k*k) / (2*sigma*sigma));
        h_kernel[center + k] = val;
        sum += val;
    }
    for (int k = 0; k < size; k++) {
        h_kernel[k] /= sum;
    }
    
    // 分配设备内存
    float *d_input, *d_temp, *d_output, *d_kernel;
    hipMalloc(&d_input, img.width * img.height * sizeof(float));
    hipMalloc(&d_temp, img.width * img.height * sizeof(float));
    hipMalloc(&d_output, img.width * img.height * sizeof(float));
    hipMalloc(&d_kernel, size * sizeof(float));
    
    // 复制数据到设备
    hipMemcpy(d_input, img.data, img.width * img.height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel.data(), size * sizeof(float), hipMemcpyHostToDevice);
    
    // 设置grid和block大小
    dim3 block(16, 16);
    dim3 grid((img.width + block.x - 1) / block.x, 
              (img.height + block.y - 1) / block.y);
    
    // 执行水平和垂直方向的卷积
    gaussian_blur_horizontal<<<grid, block>>>(d_input, d_temp, d_kernel,
                                            img.width, img.height, size, center);
    gaussian_blur_vertical<<<grid, block>>>(d_temp, d_output, d_kernel,
                                          img.width, img.height, size, center);
    
    // 创建输出图像
    Image result(img.width, img.height, 1);
    
    // 复制结果回主机
    hipMemcpy(result.data, d_output, img.width * img.height * sizeof(float), hipMemcpyDeviceToHost);
    
    // 清理设备内存
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_output);
    hipFree(d_kernel);
    
    return result;
} 


__global__ void compute_gradient_kernel(const float *input, float *output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        float gx = 0.5f * (input[y*width + (x+1)] - input[y*width + (x-1)]);
        float gy = 0.5f * (input[(y+1)*width + x] - input[(y-1)*width + x]);

        // output前一半存gx，后一半存gy
        output[y*width + x] = gx;  
        output[width*height + y*width + x] = gy;
    } else if (x < width && y < height) {
        // 边界处可设为0或保持默认值
        output[y*width + x] = 0.0f;
        output[width*height + y*width + x] = 0.0f;
    }
}


__global__ void dog_kernel(const float* img1, const float* img2, float* out, int pixels) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if (idx < pixels) {
        out[idx] = img2[idx]-img1[idx];
    }
}

namespace sift {

ScaleSpacePyramid generate_gradient_pyramid_cuda(const ScaleSpacePyramid& pyramid)
{
    ScaleSpacePyramid grad_pyramid = {
        pyramid.num_octaves,
        pyramid.imgs_per_octave,
        std::vector<std::vector<Image>>(pyramid.num_octaves)
    };

    // 为每张图像分配GPU内存进行处理
    for (int i = 0; i < pyramid.num_octaves; i++) {
        grad_pyramid.octaves[i].reserve(pyramid.imgs_per_octave);
        for (int j = 0; j < pyramid.imgs_per_octave; j++) {
            const Image& in_img = pyramid.octaves[i][j];
            int width = in_img.width;
            int height = in_img.height;
            assert(in_img.channels == 1);

            Image grad(width, height, 2);

            float *d_input, *d_output;
            size_t img_size = width * height * sizeof(float);
            hipMalloc(&d_input, img_size);
            hipMalloc(&d_output, img_size * 2); // 两个通道

            // 复制输入数据到GPU
            hipMemcpy(d_input, in_img.data, img_size, hipMemcpyHostToDevice);

            dim3 block(32,8);
            dim3 grid((width + block.x - 1)/block.x, (height + block.y - 1)/block.y);

            compute_gradient_kernel<<<grid, block>>>(d_input, d_output, width, height);
            hipDeviceSynchronize();

            // 拷贝结果回CPU
            hipMemcpy(grad.data, d_output, img_size*2, hipMemcpyDeviceToHost);

            // 释放GPU内存
            hipFree(d_input);
            hipFree(d_output);

            grad_pyramid.octaves[i].push_back(grad);
        }
    }

    return grad_pyramid;
}



ScaleSpacePyramid generate_dog_pyramid_cuda(const ScaleSpacePyramid& gauss_pyr) {
    ScaleSpacePyramid dog_pyr = {
        gauss_pyr.num_octaves,
        gauss_pyr.imgs_per_octave - 1,
        std::vector<std::vector<Image>>(gauss_pyr.num_octaves)
    };

    for(int i=0; i<gauss_pyr.num_octaves; i++){
        dog_pyr.octaves[i].reserve(dog_pyr.imgs_per_octave);
        for (int j=1; j<gauss_pyr.imgs_per_octave; j++){
            const Image& img1 = gauss_pyr.octaves[i][j-1];
            const Image& img2 = gauss_pyr.octaves[i][j];
            assert(img1.channels == 1 && img2.channels == 1);
            assert(img1.width == img2.width && img1.height == img2.height);

            int pixels = img1.width * img1.height;
            size_t sz = pixels*sizeof(float);
            float *d_img1, *d_img2, *d_out;
            hipMalloc(&d_img1, sz);
            hipMalloc(&d_img2, sz);
            hipMalloc(&d_out, sz);

            hipMemcpy(d_img1, img1.data, sz, hipMemcpyHostToDevice);
            hipMemcpy(d_img2, img2.data, sz, hipMemcpyHostToDevice);

            dim3 block(256);
            dim3 grid((pixels+255)/256);
            dog_kernel<<<grid,block>>>(d_img1,d_img2,d_out,pixels);
            hipDeviceSynchronize();

            Image diff(img1.width, img1.height, 1);
            hipMemcpy(diff.data, d_out, sz, hipMemcpyDeviceToHost);

            dog_pyr.octaves[i].push_back(diff);

            hipFree(d_img1);
            hipFree(d_img2);
            hipFree(d_out);
        }
    }
    return dog_pyr;
}

__global__ void check_contrast_and_extremum_kernel(const float* dog_image, const float* prev_image, 
    const float* next_image, int width, int height, int* potential_keypoints, 
    int* counter, float contrast_thresh) {
    
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (x >= width-1 || y >= height-1) return;
    
    int idx = y * width + x;
    float val = dog_image[idx];
    
    // Early contrast threshold check
    if (abs(val) < 0.8f * contrast_thresh) return;
    
    bool is_min = true, is_max = true;
    
    // Check against current, previous and next DoG images
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            int curr_idx = (y + dy) * width + (x + dx);
            
            float prev_val = prev_image[curr_idx];
            float next_val = next_image[curr_idx];
            float curr_val = dog_image[curr_idx];
            
            if (prev_val > val || next_val > val || 
                (curr_val > val && !(dx == 0 && dy == 0))) {
                is_max = false;
            }
            if (prev_val < val || next_val < val || 
                (curr_val < val && !(dx == 0 && dy == 0))) {
                is_min = false;
            }
            
            if (!is_min && !is_max) return;
        }
    }
    
    if (is_min || is_max) {
        int insert_idx = atomicAdd(counter, 1);
        potential_keypoints[insert_idx * 3] = x;
        potential_keypoints[insert_idx * 3 + 1] = y;
        potential_keypoints[insert_idx * 3 + 2] = (is_max ? 1 : -1); // Store extremum type
    }
}

std::vector<Keypoint> find_keypoints_cuda(const std::vector<std::vector<Image>>& dog_octaves,
                                         float contrast_thresh, float edge_thresh) {
    std::vector<Keypoint> tmp_kps;
    
    // Allocate device memory for counter
    int* d_counter;
    hipMalloc(&d_counter, sizeof(int));
    
    for (int i = 0; i < dog_octaves.size(); i++) {
        const std::vector<Image>& octave = dog_octaves[i];
        for (int j = 1; j < octave.size()-1; j++) {
            const Image& curr_img = octave[j];
            const Image& prev_img = octave[j-1];
            const Image& next_img = octave[j+1];
            
            int width = curr_img.width;
            int height = curr_img.height;
            
            // Allocate device memory
            float *d_curr, *d_prev, *d_next;
            hipMalloc(&d_curr, width * height * sizeof(float));
            hipMalloc(&d_prev, width * height * sizeof(float));
            hipMalloc(&d_next, width * height * sizeof(float));
            
            // Copy data to device
            hipMemcpy(d_curr, curr_img.data, width * height * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_prev, prev_img.data, width * height * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_next, next_img.data, width * height * sizeof(float), hipMemcpyHostToDevice);
            
            // Reset counter
            hipMemset(d_counter, 0, sizeof(int));
            
            // Allocate memory for potential keypoints (x, y, extremum_type)
            int max_keypoints = width * height; // Maximum possible keypoints
            int* d_potential_keypoints;
            hipMalloc(&d_potential_keypoints, max_keypoints * 3 * sizeof(int));
            
            // Launch kernel
            dim3 block(16, 16);
            dim3 grid((width + block.x - 1) / block.x, 
                     (height + block.y - 1) / block.y);
            
            check_contrast_and_extremum_kernel<<<grid, block>>>(
                d_curr, d_prev, d_next, width, height,
                d_potential_keypoints, d_counter, contrast_thresh);
            
            // Get number of detected keypoints
            int num_keypoints;
            hipMemcpy(&num_keypoints, d_counter, sizeof(int), hipMemcpyDeviceToHost);
            
            if (num_keypoints > 0) {
                // Allocate host memory for keypoint coordinates
                std::vector<int> keypoint_data(num_keypoints * 3);
                hipMemcpy(keypoint_data.data(), d_potential_keypoints, 
                          num_keypoints * 3 * sizeof(int), hipMemcpyDeviceToHost);
                
                // Process keypoints
                for (int k = 0; k < num_keypoints; k++) {
                    int x = keypoint_data[k * 3];
                    int y = keypoint_data[k * 3 + 1];
                    
                    Keypoint kp = {x, y, i, j, -1, -1, -1, -1};
                    bool kp_is_valid = refine_or_discard_keypoint(kp, octave, contrast_thresh, edge_thresh);
                    if (kp_is_valid) {
                        tmp_kps.push_back(kp);
                    }
                }
            }
            
            // Cleanup
            hipFree(d_curr);
            hipFree(d_prev);
            hipFree(d_next);
            hipFree(d_potential_keypoints);
        }
    }
    
    hipFree(d_counter);
    return tmp_kps;
}

} // namespace sift

__global__ void smooth_histogram_kernel(float* hist, float* tmp_hist, int n_bins) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_bins) {
        int prev_idx = (idx - 1 + n_bins) % n_bins;
        int next_idx = (idx + 1) % n_bins;
        tmp_hist[idx] = (hist[prev_idx] + hist[idx] + hist[next_idx]) / 3.0f;
    }
}

void smooth_histogram_cuda(float* hist, int n_bins) {
    float *d_hist, *d_tmp_hist;
    
    // Allocate device memory
    hipMalloc(&d_hist, n_bins * sizeof(float));
    hipMalloc(&d_tmp_hist, n_bins * sizeof(float));
    
    // Copy histogram to device
    hipMemcpy(d_hist, hist, n_bins * sizeof(float), hipMemcpyHostToDevice);
    
    // Calculate grid and block dimensions
    int block_size = 256;
    int grid_size = (n_bins + block_size - 1) / block_size;
    
    // Perform 6 iterations of smoothing
    for (int i = 0; i < 6; i++) {
        smooth_histogram_kernel<<<grid_size, block_size>>>(d_hist, d_tmp_hist, n_bins);
        hipMemcpy(d_hist, d_tmp_hist, n_bins * sizeof(float), hipMemcpyDeviceToDevice);
    }
    
    // Copy result back to host
    hipMemcpy(hist, d_hist, n_bins * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_hist);
    hipFree(d_tmp_hist);
}
